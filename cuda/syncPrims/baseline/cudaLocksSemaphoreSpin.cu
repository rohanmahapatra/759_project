#include "hip/hip_runtime.h"
#ifndef __CUDALOCKSSEMAPHORESPIN_CU__
#define __CUDALOCKSSEMAPHORESPIN_CU__

#include "cudaLocks.h"

inline __host__ hipError_t cudaSemaphoreCreateSpin(cudaSemaphore_t * const handle,
                                                    const int semaphoreNumber,
                                                    const unsigned int count,
                                                    const int NUM_SM)
{
  // Here we set the initial value to be count+1, this allows us to do an
  // atomicExch(sem, 0) and basically use the semaphore value as both a
  // lock and a semaphore.
  unsigned int initialValue = (count + 1), zero = 0;
  *handle = semaphoreNumber;
  for (int id = 0; id < NUM_SM; ++id) { // need to set these values for all SMs
    hipMemcpy(&(cpuLockData->semaphoreBuffers[((semaphoreNumber * 4 * NUM_SM) + (id * 4))]), &initialValue, sizeof(initialValue), hipMemcpyHostToDevice);
    hipMemcpy(&(cpuLockData->semaphoreBuffers[((semaphoreNumber * 4 * NUM_SM) + (id * 4)) + 1]), &zero, sizeof(zero), hipMemcpyHostToDevice);
    hipMemcpy(&(cpuLockData->semaphoreBuffers[((semaphoreNumber * 4 * NUM_SM) + (id * 4)) + 2]), &zero, sizeof(zero), hipMemcpyHostToDevice);
    hipMemcpy(&(cpuLockData->semaphoreBuffers[((semaphoreNumber * 4 * NUM_SM) + (id * 4)) + 3]), &initialValue, sizeof(initialValue), hipMemcpyHostToDevice);
  }
  return hipSuccess;
}

inline __device__ bool cudaSemaphoreSpinTryWait(const cudaSemaphore_t sem,
                                                const bool isWriter,
                                                const unsigned int maxSemCount,
                                                unsigned int * semaphoreBuffers,
                                                const int NUM_SM)
{
  const bool isMasterThread = (threadIdx.x == 0 && threadIdx.y == 0 &&
                               threadIdx.z == 0);
  /*
    Each sem has NUM_SM * 4 locations in the buffer.  Of these locations, each
    SM uses 4 of them (current count, head, tail, max count).  For the global
    semaphore all SMs use semaphoreBuffers[sem * 4 * NUM_SM].
  */
  unsigned int * const currCount = semaphoreBuffers + (sem * 4 * NUM_SM);
  unsigned int * const lock = currCount + 1;
  /*
    Reuse the tail for the "writers are waiting" flag since tail is unused.

    For now just use to indicate that at least 1 writer is waiting instead of
    a count to make sure that readers aren't totally starved out until all the
    writers are done.
  */
  unsigned int * const writerWaiting = currCount + 2;
  __shared__ bool acq1, acq2;

  __syncthreads();
  if (isMasterThread)
  {
    acq1 = false;
    // try to acquire the sem head "lock"
    if (atomicCAS(lock, 0, 1) == 0) {
      // atomicCAS acts as a load acquire, need TF to enforce ordering
      __threadfence();
      acq1 = true;
    }
  }
  __syncthreads();

  if (!acq1) { return false; } // return if we couldn't acquire the lock
  if (isMasterThread)
  {
    acq2 = false;
    /*
      NOTE: currCount is only accessed by 1 TB at a time and has a lock around
      it, so we can safely access it as a regular data access instead of with
      atomics.
    */
    unsigned int currSemCount = currCount[0];

    if (isWriter) {
      // writer needs the count to be == maxSemCount to enter the critical
      // section (otherwise there are readers in the critical section)
      if (currSemCount == maxSemCount) { acq2 = true; }
    } else {
      // if there is a writer waiting, readers aren't allowed to enter the
      // critical section
      if (writerWaiting[0] == 0) {
        // readers need count > 1 to enter critical section (otherwise semaphore
        // is full)
        if (currSemCount > 1) { acq2 = true; }
      }
    }
  }
  __syncthreads();

  if (!acq2) // release the sem head "lock" since the semaphore was full
  {
    // writers set a flag to note that they are waiting so more readers don't
    // join after the writer started waiting
    if (isWriter) { writerWaiting[0] = 1; /* if already 1, just reset to 1 */ }

    if (isMasterThread) {
      // atomicExch acts as a store release, need TF to enforce ordering
      __threadfence();
      atomicExch(lock, 0);
    }
    __syncthreads();
    return false;
  }
  __syncthreads();

  if (isMasterThread) {
    /*
      NOTE: currCount is only accessed by 1 TB at a time and has a lock around
      it, so we can safely access it as a regular data access instead of with
      atomics.
    */
    if (isWriter) {
      /*
        writer decrements the current count of the semaphore by the max to
        ensure that no one else can enter the critical section while it's
        writing.
      */
      currCount[0] -= maxSemCount;

      // writers also need to unset the "writer is waiting" flag
      writerWaiting[0] = 0;
    } else {
      // readers decrement the current count of the semaphore by 1 so other
      // readers can also read the data (but not the writers since they needs
      // the entire CS).
      --currCount[0];
    }

    // atomicExch acts as a store release, need TF to enforce ordering
    __threadfence();
    // now that we've updated the semaphore count can release the lock
    atomicExch(lock, 0);
  }
  __syncthreads();

  return true;
}

inline __device__ void cudaSemaphoreSpinWait(const cudaSemaphore_t sem,
                                             const bool isWriter,
                                             const unsigned int maxSemCount,
                                             unsigned int * semaphoreBuffers,
                                             const int NUM_SM)
{
  while (!cudaSemaphoreSpinTryWait(sem, isWriter, maxSemCount, semaphoreBuffers, NUM_SM))
  {
    __syncthreads();
  }
}

inline __device__ void cudaSemaphoreSpinPost(const cudaSemaphore_t sem,
                                             const bool isWriter,
                                             const unsigned int maxSemCount,
                                             unsigned int * semaphoreBuffers,
                                             const int NUM_SM)
{
  const bool isMasterThread = (threadIdx.x == 0 && threadIdx.y == 0 &&
                               threadIdx.z == 0);
  /*
    Each sem has NUM_SM * 4 locations in the buffer.  Of these locations, each
    SM uses 4 of them (current count, head, tail, max count).  For the global
    semaphore use semaphoreBuffers[sem * 4 * NUM_SM].
  */
  unsigned int * const currCount = semaphoreBuffers + (sem * 4 * NUM_SM);
  unsigned int * const lock = currCount + 1;
  __shared__ bool acquired;

  if (isMasterThread) { acquired = false; }
  __syncthreads();

  while (!acquired)
  {
    if (isMasterThread)
    {
      /*
        NOTE: This CAS will trigger an invalidation since we overload CAS's.
        Since most of the data in the local critical section is written, it
        hopefully won't affect performance too much.
      */
      // try to acquire sem head lock
      if (atomicCAS(lock, 0, 1) == 0) {
        // atomicCAS acts as a load acquire, need TF to enforce ordering
        __threadfence();
        acquired = true;
      }
      else                            { acquired = false; }
    }
    __syncthreads();
  }
  __syncthreads();

  if (isMasterThread) {
    /*
      NOTE: currCount is only accessed by 1 TB at a time and has a lock around
      it, so we can safely access it as a regular data access instead of with
      atomics.
    */
    if (isWriter) {
      // writers add the max value to the semaphore to allow the readers to
      // start accessing the critical section.
      currCount[0] += maxSemCount;
    } else {
      // readers add 1 to the semaphore
      ++currCount[0];
    }

    // atomicExch acts as a store release, need TF to enforce ordering
    __threadfence();
    // now that we've updated the semaphore count can release the lock
    atomicExch(lock, 0);
  }
  __syncthreads();
}

// same wait algorithm but with local scope and per-SM synchronization
inline __device__ bool cudaSemaphoreSpinTryWaitLocal(const cudaSemaphore_t sem,
                                                     const unsigned int smID,
                                                     const bool isWriter,
                                                     const unsigned int maxSemCount,
                                                     unsigned int * semaphoreBuffers,
                                                     const int NUM_SM)
{
  const bool isMasterThread = (threadIdx.x == 0 && threadIdx.y == 0 &&
                               threadIdx.z == 0);
  // Each sem has NUM_SM * 4 locations in the buffer.  Of these locations, each
  // SM gets 4 of them (current count, head, tail, max count).  So SM 0 starts
  // at semaphoreBuffers[sem * 4 * NUM_SM].
  unsigned int * const currCount = semaphoreBuffers +
                                       ((sem * 4 * NUM_SM) + (smID * 4));
  unsigned int * const lock = currCount + 1;
  /*
    Reuse the tail for the "writers are waiting" flag since tail is unused.

    For now just use to indicate that at least 1 writer is waiting instead of
    a count to make sure that readers aren't totally starved out until all the
    writers are done.
  */
  unsigned int * const writerWaiting = currCount + 2;
  __shared__ bool acq1, acq2;

  __syncthreads();
  if (isMasterThread)
  {
    acq1 = false;
    // try to acquire the sem head "lock"
    if (atomicCAS(lock, 0, 1) == 0) {
      // atomicCAS acts as a load acquire, need TF to enforce ordering locally
      __threadfence_block();
      acq1 = true;
    }
  }
  __syncthreads();

  if (!acq1) { return false; } // return if we couldn't acquire the lock
  if (isMasterThread)
  {
    acq2 = false;
    /*
      NOTE: currCount is only accessed by 1 TB at a time and has a lock around
      it, so we can safely access it as a regular data access instead of with
      atomics.
    */
    unsigned int currSemCount = currCount[0];

    if (isWriter) {
      // writer needs the count to be == maxSemCount to enter the critical
      // section (otherwise there are readers in the critical section)
      if (currSemCount == maxSemCount) { acq2 = true; }
    } else {
      // if there is a writer waiting, readers aren't allowed to enter the
      // critical section
      if (writerWaiting[0] == 0) {
        // readers need count > 1 to enter critical section (otherwise semaphore
        // is full)
        if (currSemCount > 1) { acq2 = true; }
      }
    }
  }
  __syncthreads();

  if (!acq2) // release the sem head "lock" since the semaphore was full
  {
    // writers set a flag to note that they are waiting so more readers don't
    // join after the writer started waiting
    if (isWriter) { writerWaiting[0] = 1; /* if already 1, just reset to 1 */ }

    if (isMasterThread) {
      // atomicExch acts as a store release, need TF to enforce ordering locally
      __threadfence_block();
      atomicExch(lock, 0);
    }
    __syncthreads();
    return false;
  }
  __syncthreads();

  if (isMasterThread) {
    /*
      NOTE: currCount is only accessed by 1 TB at a time and has a lock around
      it, so we can safely access it as a regular data access instead of with
      atomics.
     */
    if (isWriter) {
      /*
        writer decrements the current count of the semaphore by the max to
        ensure that no one else can enter the critical section while it's
        writing.
      */
      currCount[0] -= maxSemCount;

      // writers also need to unset the "writer is waiting" flag
      writerWaiting[0] = 0;
    } else {
      /*
        readers decrement the current count of the semaphore by 1 so other
        readers can also read the data (but not the writers since they needs
        the entire CS).
      */
      --currCount[0];
    }

    // atomicExch acts as a store release, need TF to enforce ordering locally
    __threadfence_block();
    // now that we've updated the semaphore count can release the lock
    atomicExch(lock, 0);
  }
  __syncthreads();

  return true;
}

inline __device__ void cudaSemaphoreSpinWaitLocal(const cudaSemaphore_t sem,
                                                  const unsigned int smID,
                                                  const bool isWriter,
                                                  const unsigned int maxSemCount,
                                                  unsigned int * semaphoreBuffers,
                                                  const int NUM_SM)
{
  while (!cudaSemaphoreSpinTryWaitLocal(sem, smID, isWriter, maxSemCount, semaphoreBuffers, NUM_SM))
  {
    __syncthreads();
  }
}

inline __device__ void cudaSemaphoreSpinPostLocal(const cudaSemaphore_t sem,
                                                  const unsigned int smID,
                                                  const bool isWriter,
                                                  const unsigned int maxSemCount,
                                                  unsigned int * semaphoreBuffers,
                                                  const int NUM_SM)
{
  bool isMasterThread = (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0);
  // Each sem has NUM_SM * 4 locations in the buffer.  Of these locations, each
  // SM gets 4 of them.  So SM 0 starts at semaphoreBuffers[sem * 4 * NUM_SM].
  unsigned int * const currCount = semaphoreBuffers +
                                       ((sem * 4 * NUM_SM) + (smID * 4));
  unsigned int * const lock = currCount + 1;
  __shared__ bool acquired;

  if (isMasterThread) { acquired = false; }
  __syncthreads();

  while (!acquired)
  {
    if (isMasterThread)
    {
      /*
        NOTE: This CAS will trigger an invalidation since we overload CAS's.
        Since most of the data in the local critical section is written, it
        hopefully won't affect performance too much.
      */
      // try to acquire sem head lock
      if (atomicCAS(lock, 0, 1) == 0) {
        // atomicCAS acts as a load acquire, need TF to enforce ordering locally
        __threadfence_block();
        acquired = true;
      }
      else                            { acquired = false; }
    }
    __syncthreads();
  }
  __syncthreads();

  if (isMasterThread) {
    /*
      NOTE: currCount is only accessed by 1 TB at a time and has a lock around
      it, so we can safely access it as a regular data access instead of with
      atomics.
    */
    if (isWriter) {
      // writers add the max value to the semaphore to allow the readers to
      // start accessing the critical section.
      currCount[0] += maxSemCount;
    } else {
      // readers add 1 to the semaphore
      ++currCount[0];
    }

    // atomicExch acts as a store release, need TF to enforce ordering locally
    __threadfence_block();
    // now that we've updated the semaphore count can release the lock
    atomicExch(lock, 0);
  }
  __syncthreads();
}

#endif // #ifndef __CUDALOCKSSEMAPHORESPIN_CU__
