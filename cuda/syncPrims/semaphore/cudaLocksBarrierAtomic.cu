#include "hip/hip_runtime.h"
#ifndef __CUDALOCKSBARRIERATOMIC_CU__
#define __CUDALOCKSBARRIERATOMIC_CU__

#include "cudaLocks.h"

inline __device__ void cudaBarrierAtomicSub(unsigned int * globalBarr,
                                            int * done,
                                            // numBarr represents the number of
                                            // TBs going to the barrier
                                            const unsigned int numBarr,
                                            int backoff,
                                            const bool isMasterThread, bool *local_sense1, bool *global_sense)
{
  __syncthreads();
  bool s = ~local_sense1[blockIdx.x];
  local_sense1[blockIdx.x] = s;
  if (isMasterThread)
  {
    //bool s = ~local_sense1[blockIdx.x];
    //local_sense1[blockIdx.x] = s;
  
    //*done = 0;
    // atomicInc acts as a store release, need TF to enforce ordering locally
    __threadfence_block();
    /*
      atomicInc effectively adds 1 to atomic for each TB that's part of the
      barrier.  For the local barrier, this requires using the per-CU
      locations.
    */
    atomicInc(globalBarr, 0x7FFFFFFF);
  }
  __syncthreads();

  //while (global_sense != s && intermediate_sense != s)
  while (*global_sense != s)
  {
    if (isMasterThread)
    {
      /*
        Once all of the TBs on this SM have incremented the value at atomic,
        then the value (for the local barrier) should be equal to the # of TBs
        on this SM.  Once that is true, then we want to reset the atomic to 0
        and proceed because all of the TBs on this SM have reached the local
        barrier.
      */
      if (atomicCAS(globalBarr, numBarr, 0) == 0) {
        // atomicCAS acts as a load acquire, need TF to enforce ordering
        // locally
        __threadfence_block();
        //*last_block = blockIdx.x;
        *global_sense = s;
      }
      else { // increase backoff to avoid repeatedly hammering global barrier
             // (capped) exponential backoff
               backoff = (((backoff << 1) + 1) & (MAX_BACKOFF-1));
            }
       
    }
    __syncthreads();
   // do exponential backoff to reduce the number of times we pound the global
    // barrier
    if (!*done) {
      for (int i = 0; i < backoff; ++i) { ; }
      __syncthreads();
    }
  
  }
}

inline __device__ void cudaBarrierAtomic(unsigned int * barrierBuffers,
                                         // numBarr represents the number of
                                         // TBs going to the barrier
                                         const unsigned int numBarr,
                                         const bool isMasterThread, bool *global_sense)
{
  unsigned int * atomic1 = barrierBuffers;
  //unsigned int * atomic2 = atomic1 + 1;
  __shared__ int done1; // done2;
  __shared__ int backoff;

  if (isMasterThread) {
    backoff = 1;
  }
  __syncthreads();

  //bool local_sense1[numBarr];
  bool local_sense1[1025];
  for (int i = 0; i<numBarr; i++)
    local_sense1[i] = true;  


  cudaBarrierAtomicSub(atomic1, &done1, numBarr, backoff, isMasterThread, local_sense1, global_sense);
  // second barrier is necessary to provide a facesimile for a sense-reversing
  // barrier
  //cudaBarrierAtomicSub(atomic2, &done2, numBarr, backoff, isMasterThread);
}

// does local barrier amongst all of the TBs on an SM
inline __device__ void cudaBarrierAtomicSubLocal(unsigned int * perSMBarr,
                                                 int * done,
                                                 const unsigned int numTBs_thisSM,
                                                 const bool isMasterThread, bool *local_sense, int *last_block, bool * sense)
{
  __syncthreads();
  bool s = ~local_sense[blockIdx.x];
  local_sense[blockIdx.x] = s;
  if (isMasterThread)
  {
   
  
    //*done = 0;
    // atomicInc acts as a store release, need TF to enforce ordering locally
    __threadfence_block();
    /*
      atomicInc effectively adds 1 to atomic for each TB that's part of the
      barrier.  For the local barrier, this requires using the per-CU
      locations.
    */
    atomicInc(perSMBarr, 0x7FFFFFFF);
  }
  __syncthreads();

  //while (global_sense != s && intermediate_sense != s)
  while (s != *sense)
  {
    if (isMasterThread)
    {
      /*
        Once all of the TBs on this SM have incremented the value at atomic,
        then the value (for the local barrier) should be equal to the # of TBs
        on this SM.  Once that is true, then we want to reset the atomic to 0
        and proceed because all of the TBs on this SM have reached the local
        barrier.
      */
      if (atomicCAS(perSMBarr, numTBs_thisSM, 0) == 0) {
        // atomicCAS acts as a load acquire, need TF to enforce ordering
        // locally
        __threadfence_block();
        *last_block = blockIdx.x;
        *sense = s;
      }
    }
    __syncthreads();
  }
}

// does local barrier amongst all of the TBs on an SM
inline __device__ void cudaBarrierAtomicLocal(unsigned int * perSMBarrierBuffers,
                                              const unsigned int smID,
                                              const unsigned int numTBs_thisSM,
                                              const bool isMasterThread,
                                              const int MAX_BLOCKS, int *last_block, bool *sense)
{
  // each SM has MAX_BLOCKS locations in barrierBuffers, so my SM's locations
  // start at barrierBuffers[smID*MAX_BLOCKS]
  unsigned int * atomic1 = perSMBarrierBuffers + (smID * MAX_BLOCKS);


  //unsigned int * atomic2 = atomic1 + 1;
  __shared__ int done1; //, done2;
  //printf("Atomic 1: %d \n", atomic1);
  
  //bool local_sense[numTBs_thisSM];
  bool local_sense[1025];
  
  for (int i = 0; i<numTBs_thisSM; i++)
    local_sense[i] = true;  


  cudaBarrierAtomicSubLocal(atomic1, &done1, numTBs_thisSM, isMasterThread, local_sense, last_block, sense);
  // second barrier is necessary to approproximate a sense-reversing barrier
  //cudaBarrierAtomicSubLocal(atomic2, &done2, numTBs_thisSM, isMasterThread);
}

/*
  Helper function for joining the barrier with the atomic tree barrier.
*/
__device__ void joinBarrier_helper(unsigned int * barrierBuffers,
                                   unsigned int * perSMBarrierBuffers,
                                   const unsigned int numBlocksAtBarr,
                                   const int smID,
                                   const int perSM_blockID,
                                   const int numTBs_perSM,
                                   const bool isMasterThread,
                                   const int MAX_BLOCKS) {

  int last_block = blockIdx.x;  // by default
  bool sense = true;
  bool global_sense = true;
  if (numTBs_perSM > 1) {
    cudaBarrierAtomicLocal(perSMBarrierBuffers, smID, numTBs_perSM,isMasterThread, MAX_BLOCKS, &last_block, &sense);

    
    // only 1 TB per SM needs to do the global barrier since we synchronized
    // the TBs locally first
    if (threadIdx.x == last_block) {
      cudaBarrierAtomic(barrierBuffers, numBlocksAtBarr, isMasterThread, &global_sense);
    }

    // all TBs on this SM do a local barrier to ensure global barrier is
    // reached
    while (sense != global_sense)
    {
      ;
    }
    //__threadfence_block;   
    //cudaBarrierAtomicLocal(perSMBarrierBuffers, smID, numTBs_perSM,
                          // isMasterThread, MAX_BLOCKS, last_block);
  } else { // if only 1 TB on the SM, no need for the local barriers
    cudaBarrierAtomic(barrierBuffers, numBlocksAtBarr, isMasterThread, &global_sense);
    //cudaBarrierAtomic(barrierBuffers, numBlocksAtBarr, isMasterThread);
  }
}

#endif
